
#include <hip/hip_runtime.h>
// CUDA kernel for matrix multiplication (GEMM)
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBColumns)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numARows && col < numBColumns)
    {
        float sum = 0.0f;
        for (int i = 0; i < numAColumns; ++i)
        {
            sum += A[row * numAColumns + i] * B[i * numBColumns + col];
        }
        C[row * numBColumns + col] = sum;
    }
}